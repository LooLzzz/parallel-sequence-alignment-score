#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "cudaFunctions.cuh"

__global__ void _compute(TASK *tasks, int tasks_count);
__device__ void computeSigns(TASK *tasks);
__device__ void computeScore(TASK *tasks);

__device__ int _strlen(const char *str);
__device__ char *_strchr(const char *str, char ch);


int computeTasks(TASK *tasks, int tasks_count, DIR dir)
{
    hipError_t err;
    int size, blocksPerGrid;
    TASK *device_tasks;
          
    size = tasks_count * sizeof(TASK);
    hipMalloc((void**) &device_tasks, size);
    cudaCheckErr();
    hipMemcpy(device_tasks, tasks, size, hipMemcpyHostToDevice);
    cudaCheckErr();

    blocksPerGrid  = (tasks_count + threadsPerBlock - 1) / threadsPerBlock;
    _compute<<<blocksPerGrid, threadsPerBlock>>>(device_tasks, tasks_count);
    cudaCheckErr();
    
    hipMemcpy(tasks, device_tasks, size, hipMemcpyDeviceToHost);
    cudaCheckErr();
    
    hipFree(device_tasks);
    cudaCheckErr();

    return hipSuccess;
}

__global__ void _compute(TASK *tasks, int tasks_count)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (idx > tasks_count-1)
        return; 
        
    computeSigns(tasks);
    computeScore(tasks);
}

__device__ void computeSigns(TASK *tasks)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    // TASK *task = tasks + i;

    char ConservativeGroups[CONSERVATIVE_GROUPS_COUNT][GROUP_MAX] = {
        "NDEQ", "NEQK", "STA",
        "MILV", "QHRK", "NHQK",
        "FYW", "HY", "MILF"
    };
    char SemiConservativeGroups[SEMI_CONSERVATIVE_GROUPS_COUNT][GROUP_MAX] = {
        "SAG", "ATV", "CSA",
        "SGND", "STPA", "STNK",
        "NEQHRK", "NDEQHK", "SNDEQK",
        "HFY", "FVLIM"
    };

    char *signs = tasks[idx].signs;
    char *seq1 = tasks[idx].seq1 + tasks[idx].offset;
    char *seq2 = tasks[idx].seq2;
    int seq2_len = _strlen(seq2);    
    int i, j, flag;
    char *a, *b;

    for (i = 0; i < seq2_len; i++)
    {
        // equal
        if (seq1[i] == seq2[i])
        {
            *(signs++) = '*';
            // printf("> %d > got '*' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }
        
        // conservative
        flag = FALSE;
        for (j = 0; j < CONSERVATIVE_GROUPS_COUNT; j++)
        {
            a = _strchr(ConservativeGroups[j], seq1[i]);
            b = _strchr(ConservativeGroups[j], seq2[i]);
            flag = a && b;
            if (flag)
                break;
        }
        if (flag)
        {
            *(signs++) = ':';
            // printf("> %d > got ':' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }

        // semi-conservative
        flag = FALSE;
        for (j = 0; j < SEMI_CONSERVATIVE_GROUPS_COUNT; j++)
        {
            a = _strchr(SemiConservativeGroups[j], seq1[i]);
            b = _strchr(SemiConservativeGroups[j], seq2[i]);
            flag = a && b;
            if (flag)
                break;
        }
        if (flag)
        {
            *(signs++) = '.';
            // printf("> %d > got '.' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }
        
        // none
        *(signs++) = ' ';
    }
}

__device__ void computeScore(TASK *tasks)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    char *signs = tasks[idx].signs;
    float *weights = tasks[idx].weights;
    
    tasks[idx].score = 0;
    while (*signs)
    {
        switch (*(signs++))
        {
            case '*':
                tasks[idx].score += weights[0];
                break;
            
            case ':':
                tasks[idx].score -= weights[1];
                break;
            
            case '.':
                tasks[idx].score -= weights[2];
                break;
            
            case ' ':
            default:
                tasks[idx].score -= weights[3];
                break;
        }
    }
}

__device__ int _strlen(const char *str)
{
    int len = 0;
    while (*str++)
        len++;
    return len;
}

__device__ char *_strchr(const char *str, char ch)
{
    while (*str != ch)
        if (!*str++)
            return 0;
    return (char *)str;
}
