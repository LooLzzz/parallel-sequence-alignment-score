#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "cudaFunctions.cuh"

void copyTasksToGPU(TASK *tasks, int tasks_count, GPU_TASK **device_tasks);
__global__ void _compute(GPU_TASK *tasks, int tasks_count);
__device__ void computeSigns(GPU_TASK *tasks);
__device__ void computeScore(GPU_TASK *tasks);

__device__ int _strlen(const char *str);
__device__ char *_strchr(const char *str, char ch);

__device__ char device_seq1[SEQ1_MAXLEN];
__device__ float device_weights[W_LEN];
hipError_t err;

int computeTasks(TASK *tasks, int tasks_count)
{
    int i, blocksPerGrid, seq2_len;
    GPU_TASK *device_tasks, t;

    seq2_len = strlen(tasks->seq2);
    copyTasksToGPU(tasks, tasks_count, &device_tasks);

    blocksPerGrid = (tasks_count + threadsPerBlock - 1) / threadsPerBlock;
    _compute<<<blocksPerGrid, threadsPerBlock>>>(device_tasks, tasks_count);

    for (i = 0; i < tasks_count; i++)
    {        
        hipMemcpy(&t, device_tasks+i, sizeof(GPU_TASK), hipMemcpyDeviceToHost);
        cudaCheckErr();

        hipMemcpy(tasks[i].signs, t.signs, seq2_len * sizeof(char), hipMemcpyDeviceToHost);
        cudaCheckErr();
        
        tasks[i].score = t.score;
    }

    return hipSuccess;
}

void copyTasksToGPU(TASK *tasks, int tasks_count, GPU_TASK **res)
{
    int seq2_len = strlen(tasks->seq2);
    GPU_TASK *device_tasks;

    device_tasks = (GPU_TASK *)malloc(tasks_count * sizeof(GPU_TASK));
    hipMalloc(res, tasks_count * sizeof(GPU_TASK));
    cudaCheckErr();

    // seq1
    hipMemcpyToSymbol(HIP_SYMBOL(device_seq1), tasks[0].seq1, SEQ1_MAXLEN * sizeof(char), 0, hipMemcpyHostToDevice);
    cudaCheckErr();

    // weights
    hipMemcpyToSymbol(HIP_SYMBOL(device_weights), tasks[0].weights, W_LEN * sizeof(float), 0, hipMemcpyHostToDevice);
    cudaCheckErr();

    for (int i = 0; i < tasks_count; i++)
    {
        // offset+score
        device_tasks[i].offset = tasks[i].offset;
        device_tasks[i].score = 0;
        
        // seq2
        hipMalloc(&(device_tasks[i].seq2), seq2_len * sizeof(char));
        cudaCheckErr();
        hipMemcpy(device_tasks[i].seq2, tasks[i].seq2, seq2_len * sizeof(char), hipMemcpyHostToDevice);
        cudaCheckErr();

        // signs
        hipMalloc(&(device_tasks[i].signs), seq2_len * sizeof(char));
        cudaCheckErr();
    }

    hipMemcpy(*res, device_tasks, tasks_count * sizeof(GPU_TASK), hipMemcpyHostToDevice);
    free(device_tasks);
}

__global__ void _compute(GPU_TASK *tasks, int tasks_count)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (idx > tasks_count-1)
        return; 
        
    computeSigns(tasks);
    computeScore(tasks);
}

__device__ void computeSigns(GPU_TASK *tasks)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
    // TASK *task = tasks + i;

    char ConservativeGroups[CONSERVATIVE_GROUPS_COUNT][GROUP_MAX] = {
        "NDEQ", "NEQK", "STA",
        "MILV", "QHRK", "NHQK",
        "FYW", "HY", "MILF"
    };
    char SemiConservativeGroups[SEMI_CONSERVATIVE_GROUPS_COUNT][GROUP_MAX] = {
        "SAG", "ATV", "CSA",
        "SGND", "STPA", "STNK",
        "NEQHRK", "NDEQHK", "SNDEQK",
        "HFY", "FVLIM"
    };

    char *signs = tasks[idx].signs;
    char *seq1 = device_seq1 + tasks[idx].offset;
    char *seq2 = tasks[idx].seq2;
    int seq2_len = _strlen(seq2);
    int i, j, flag;
    char *a, *b;

    for (i = 0; i < seq2_len; i++)
    {
        // equal
        if (seq1[i] == seq2[i])
        {
            *(signs++) = '*';
            // printf("> %d > got '*' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }
        
        // conservative
        flag = FALSE;
        for (j = 0; j < CONSERVATIVE_GROUPS_COUNT; j++)
        {
            a = _strchr(ConservativeGroups[j], seq1[i]);
            b = _strchr(ConservativeGroups[j], seq2[i]);
            flag = a && b;
            if (flag)
                break;
        }
        if (flag)
        {
            *(signs++) = ':';
            // printf("> %d > got ':' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }

        // semi-conservative
        flag = FALSE;
        for (j = 0; j < SEMI_CONSERVATIVE_GROUPS_COUNT; j++)
        {
            a = _strchr(SemiConservativeGroups[j], seq1[i]);
            b = _strchr(SemiConservativeGroups[j], seq2[i]);
            flag = a && b;
            if (flag)
                break;
        }
        if (flag)
        {
            *(signs++) = '.';
            // printf("> %d > got '.' > (%ch, %ch)\n", idx, seq1[i], seq2[i]);
            continue;
        }
        
        // none
        *(signs++) = ' ';
    }
}

__device__ void computeScore(GPU_TASK *tasks)
{
    int idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    char *signs = tasks[idx].signs;
    float *weights = device_weights;
    
    tasks[idx].score = 0;
    while (*signs)
    {
        switch (*(signs++))
        {
            case '*':
                tasks[idx].score += weights[0];
                break;
            
            case ':':
                tasks[idx].score -= weights[1];
                break;
            
            case '.':
                tasks[idx].score -= weights[2];
                break;
            
            case ' ':
            default:
                tasks[idx].score -= weights[3];
                break;
        }
    }
}

__device__ int _strlen(const char *str)
{
    int len = 0;
    while (*str++)
        len++;
    return len;
}

__device__ char *_strchr(const char *str, char ch)
{
    while (*str != ch)
        if (!*str++)
            return 0;
    return (char *)str;
}
