#include "hip/hip_runtime.h"
// #include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "cudaFunctions.h"

__global__ void heavy(float *A, int n, float *B, int K_MAX)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int k;
    float val;

    for (i = 0; i < n; i++)
    {
        B[i] = __FLT_MIN__; // smallest float value
        for (k = 0; k < K_MAX; k++)
        {
            val = __sinf( A[i] * __cosf(k) );
            B[i] = max(val, B[i]);
        }
    }
}

int gpuCompute(float *A, int n, float *B, int K_MAX)
{
    int size = n * sizeof(float);
    float *device_A, *device_B;
    hipMalloc((void**) &device_A, size);
    hipMalloc((void**) &device_B, size);
    
    hipMemcpy(device_A, A, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    heavy<<<blocksPerGrid, threadsPerBlock>>>(device_A, n, device_B, K_MAX);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(B, device_B, size, hipMemcpyDeviceToHost);
    hipFree(device_A);
    hipFree(device_B);
    return hipSuccess;
}
